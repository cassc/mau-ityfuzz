#include <cassert>
#include <chrono>
#include <functional>
#include <iomanip>
#include <iostream>
#include <stdexcept>
#include <thread>
#include <tuple>
#include <utility>
#include <vector>

#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char* const file, const int line)
{
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

template <class T>
float measure_performance(std::function<T(hipStream_t)> bound_function,
                          hipStream_t stream, int num_repeats = 100,
                          int num_warmups = 100)
{
    hipEvent_t start, stop;
    float time;

    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    for (int i{0}; i < num_warmups; ++i)
    {
        bound_function(stream);
    }

    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

    CHECK_CUDA_ERROR(hipEventRecord(start, stream));
    for (int i{0}; i < num_repeats; ++i)
    {
        bound_function(stream);
    }
    CHECK_CUDA_ERROR(hipEventRecord(stop, stream));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_LAST_CUDA_ERROR();
    CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));

    float const latency{time / num_repeats};

    return latency;
}

__global__ void float_addition(float* output, float const* input_1,
                               float const* input_2, uint32_t n)
{
    const uint32_t idx{blockDim.x * blockIdx.x + threadIdx.x};
    const uint32_t stride{blockDim.x * gridDim.x};
    for (uint32_t i{idx}; i < n; i += stride)
    {
        output[i] = input_1[i] + input_2[i];
    }
}

void launch_float_addition_non_mapped_pinned_memory(
    float* h_output, float const* h_input_1, float const* h_input_2,
    float* d_output, float* d_input_1, float* d_input_2, uint32_t n,
    hipStream_t stream)
{
    CHECK_CUDA_ERROR(hipMemcpyAsync(d_input_1, h_input_1, n * sizeof(float),
                                     hipMemcpyHostToDevice, stream));
    CHECK_CUDA_ERROR(hipMemcpyAsync(d_input_2, h_input_2, n * sizeof(float),
                                     hipMemcpyHostToDevice, stream));
    dim3 const threads_per_block{1024};
    dim3 const blocks_per_grid{32};
    float_addition<<<blocks_per_grid, threads_per_block, 0, stream>>>(
        d_output, d_input_1, d_input_2, n);
    CHECK_LAST_CUDA_ERROR();
    CHECK_CUDA_ERROR(hipMemcpyAsync(h_output, d_output, n * sizeof(float),
                                     hipMemcpyDeviceToHost, stream));
}

void launch_float_addition_mapped_pinned_memory(float* d_output,
                                                float* d_input_1,
                                                float* d_input_2, uint32_t n,
                                                hipStream_t stream)
{
    dim3 const threads_per_block{1024};
    dim3 const blocks_per_grid{32};
    float_addition<<<blocks_per_grid, threads_per_block, 0, stream>>>(
        d_output, d_input_1, d_input_2, n);
    CHECK_LAST_CUDA_ERROR();
}

void initialize_host_memory(float* h_buffer, uint32_t n, float value)
{
    for (int i{0}; i < n; ++i)
    {
        h_buffer[i] = value;
    }
}

bool verify_host_memory(float* h_buffer, uint32_t n, float value)
{
    for (int i{0}; i < n; ++i)
    {
        if (h_buffer[i] != value)
        {
            return false;
        }
    }
    return true;
}

int main()
{
    constexpr int const num_repeats{10};
    constexpr int const num_warmups{10};

    constexpr int const n{1000000};
    hipStream_t stream;
    CHECK_CUDA_ERROR(hipStreamCreate(&stream));

    float const v_input_1{1.0f};
    float const v_input_2{1.0f};
    float const v_output{0.0f};
    float const v_output_reference{v_input_1 + v_input_2};

    hipDeviceProp_t prop;
    CHECK_CUDA_ERROR(hipGetDeviceProperties(&prop, 0));
    if (!prop.canMapHostMemory)
    {
        throw std::runtime_error{"Device does not supported mapped memory."};
    }

    float *h_input_1, *h_input_2, *h_output;
    float *d_input_1, *d_input_2, *d_output;

    float *a_input_1, *a_input_2, *a_output;
    float *m_input_1, *m_input_2, *m_output;

    CHECK_CUDA_ERROR(hipHostMalloc(&h_input_1, n * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA_ERROR(hipHostMalloc(&h_input_2, n * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA_ERROR(hipHostMalloc(&h_output, n * sizeof(float), hipHostMallocDefault));

    CHECK_CUDA_ERROR(hipMalloc(&d_input_1, n * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_input_2, n * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_output, n * sizeof(float)));

    CHECK_CUDA_ERROR(
        hipHostAlloc(&a_input_1, n * sizeof(float), hipHostMallocMapped));
    CHECK_CUDA_ERROR(
        hipHostAlloc(&a_input_2, n * sizeof(float), hipHostMallocMapped));
    CHECK_CUDA_ERROR(
        hipHostAlloc(&a_output, n * sizeof(float), hipHostMallocMapped));

    CHECK_CUDA_ERROR(hipHostGetDevicePointer(&m_input_1, a_input_1, 0));
    CHECK_CUDA_ERROR(hipHostGetDevicePointer(&m_input_2, a_input_2, 0));
    CHECK_CUDA_ERROR(hipHostGetDevicePointer(&m_output, a_output, 0));

    // Verify the implementation correctness.
    initialize_host_memory(h_input_1, n, v_input_1);
    initialize_host_memory(h_input_2, n, v_input_2);
    initialize_host_memory(h_output, n, v_output);
    launch_float_addition_non_mapped_pinned_memory(
        h_output, h_input_1, h_input_2, d_output, d_input_1, d_input_2, n,
        stream);
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
    assert(verify_host_memory(h_output, n, v_output_reference));

    initialize_host_memory(a_input_1, n, v_input_1);
    initialize_host_memory(a_input_2, n, v_input_2);
    initialize_host_memory(a_output, n, v_output);
    launch_float_addition_mapped_pinned_memory(m_output, m_input_1, m_input_2,
                                               n, stream);
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
    assert(verify_host_memory(a_output, n, v_output_reference));

    // Measure latencies.
    std::function<void(hipStream_t)> function_non_mapped_pinned_memory{
        std::bind(launch_float_addition_non_mapped_pinned_memory, h_output,
                  h_input_1, h_input_2, d_output, d_input_1, d_input_2, n,
                  std::placeholders::_1)};
    std::function<void(hipStream_t)> function_mapped_pinned_memory{
        std::bind(launch_float_addition_mapped_pinned_memory, m_output,
                  m_input_1, m_input_2, n, std::placeholders::_1)};
    float const latency_non_mapped_pinned_memory{measure_performance(
        function_non_mapped_pinned_memory, stream, num_repeats, num_warmups)};
    float const latency_mapped_pinned_memory{measure_performance(
        function_mapped_pinned_memory, stream, num_repeats, num_warmups)};
    std::cout << std::fixed << std::setprecision(3)
              << "CUDA Kernel With Non-Mapped Pinned Memory Latency: "
              << latency_non_mapped_pinned_memory << " ms" << std::endl;
    std::cout << std::fixed << std::setprecision(3)
              << "CUDA Kernel With Mapped Pinned Memory Latency: "
              << latency_mapped_pinned_memory << " ms" << std::endl;

    CHECK_CUDA_ERROR(hipFree(d_input_1));
    CHECK_CUDA_ERROR(hipFree(d_input_2));
    CHECK_CUDA_ERROR(hipFree(d_output));
    CHECK_CUDA_ERROR(hipHostFree(h_input_1));
    CHECK_CUDA_ERROR(hipHostFree(h_input_2));
    CHECK_CUDA_ERROR(hipHostFree(h_output));
    CHECK_CUDA_ERROR(hipHostFree(a_input_1));
    CHECK_CUDA_ERROR(hipHostFree(a_input_2));
    CHECK_CUDA_ERROR(hipHostFree(a_output));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream));
}